#include "hip/hip_runtime.h"
#include "Helpers.h"

#include <vector>

__host__ __device__ void sortElement(size_t element, const int* src, int* dst, size_t size)
{
  int offset = 0;
  int newIndex = 0;
  for (size_t i = 0; i < size; ++i)
  {
    newIndex += src[element] > src[i];
    offset += (i < element) & (src[element] == src[i]);
  }
  dst[newIndex + offset] = src[element];
}

__host__ void sortCPU_execute(const int* src, int* dst, size_t size)
{
  for (size_t i = 0; i < size; ++i) sortElement(i, src, dst, size);
}

__global__ void sortGPU_execute(const int* src, int* dst, size_t size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) sortElement(index, src, dst, size);
}

__host__ void sortCPU_prepare(const std::vector<int>& src, std::vector<int>& dst)
{
  dst.resize(src.size());
  sortCPU_execute(&src[0], &dst[0], src.size());
}

__host__ void sortGPU_prepare(const std::vector<int>& srcCPU, std::vector<int>& dstCPU)
{
  dstCPU.resize(srcCPU.size());

  size_t byteSize = srcCPU.size() * sizeof(int);
  int* srcGPU = NULL;
  int* dstGPU = NULL;

  HANDLE_ERROR(hipMalloc(&srcGPU, byteSize));
  HANDLE_ERROR(hipMalloc(&dstGPU, byteSize));

  HANDLE_ERROR(hipMemcpy(srcGPU, &srcCPU[0], byteSize, hipMemcpyHostToDevice));

  CUDAConfig cudaConfig(srcCPU.size());
  unsigned int gridSize = cudaConfig.GetGridSize();
  unsigned int blockSize = cudaConfig.GetBlockSize();
  std::cout << "Grid Size: " << gridSize << std::endl;
  std::cout << "Block Size: " << blockSize << std::endl;
  sortGPU_execute<<<gridSize, blockSize>>>(srcGPU, dstGPU, srcCPU.size());

  HANDLE_ERROR(hipMemcpy(&dstCPU[0], dstGPU, byteSize, hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(dstGPU));
  HANDLE_ERROR(hipFree(srcGPU));
}

std::vector<int> generateVectorFromArgs(int argc, char **argv)
{
  std::vector<int> vectorGenerated;
  for (int i = 1; i < argc; ++i) vectorGenerated.push_back(atoi(argv[i]));
  return vectorGenerated;
}

int main_tp2(int argc, char **argv)
{
  std::vector<int> dst, src = generateVectorFromArgs(argc, argv);
  if (src.size() < 1)
  {
    std::cerr << "No elements to sort" << std::endl;
    return 1;
  }

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  std::cout << "Execute the CPU version" << std::endl;
  {
    hipEventRecord(start);
    sortCPU_prepare(src, dst);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
  }
  std::cout << "Elapsed time:" << elapsedTime << std::endl;
  PrintResults<std::vector<int>>(dst, "output_CPU.txt");

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount > 0)
  {
    std::cout << "Execute the GPU version" << std::endl;
    {
      hipEventRecord(start);
      sortGPU_prepare(src, dst);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      elapsedTime = 0;
      hipEventElapsedTime(&elapsedTime, start, stop);
    }
    std::cout << "Elapsed time:" << elapsedTime << std::endl;
    PrintResults<std::vector<int>>(dst, "ouput_GPU.txt");
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}